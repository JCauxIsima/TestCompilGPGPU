#include "hip/hip_runtime.h"

#include "cuda_test_compil.hpp"

#include <hip/hip_runtime.h>

#include <iostream>

namespace {

__global__ void doNothing()
{
  [[maybe_unused]] int i = blockIdx.x * blockDim.x + threadIdx.x;
  [[maybe_unused]] int j = blockIdx.y * blockDim.y + threadIdx.y;
}

} // namespace

void runOnGPU()
{
  doNothing<<<1, 10>>>();
  auto err = hipGetLastError();
  if (err != hipSuccess)
  {
    std::cerr << "Error on runOnGPU: " << hipGetErrorString(err) << std::endl;
  }
}
